#include "hip/hip_runtime.h"
//*****************************************************************************************//
//  hough.cu - CUDA Hough Transform Benchmark
//
//  Authors: Ramnarayan Krishnamurthy, University of Colorado (Shreyas.Ramnarayan@gmail.com)
//	         Matthew Demi Vis, Embry-Riddle Aeronautical University (MatthewVis@gmail.com)
//			 
//	This code was used to obtain results documented in the SPIE Sensor and Technologies paper: 
//	S. Siewert, V. Angoth, R. Krishnamurthy, K. Mani, K. Mock, S. B. Singh, S. Srivistava, 
//	C. Wagner, R. Claus, M. Demi Vis, “Software Defined Multi-Spectral Imaging for Arctic 
//	Sensor Networks”, SPIE Algorithms and Technologies for Multipectral, Hyperspectral, and 
//	Ultraspectral Imagery XXII, Baltimore, Maryland, April 2016. 
//
//	This code was developed for, tested and run on a Jetson TK1 development kit by NVIDIA
//  running Ubuntu 14.04 
//	
//	Please use at your own risk. We are sharing so that other researchers and developers can 
//	recreate our results and make suggestions to improve and extend the benchmarks over time.
//
//*****************************************************************************************//

// Standard Includes 
#include <stdio.h>
#include <stdlib.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <unistd.h>
#include <fcntl.h>
#include <errno.h>
#include <sys/io.h>
#include <iostream>

#include <time.h>
#include <pthread.h>
#include <sched.h>

// Project Includes
#include <hip/hip_runtime.h>
#include "ppm.h"
#include "options.h"

// Project-Specific Defines
#define PRIO_ADJUST 	5
#define BLOCK_SIZE_1 	8	
#define BLOCK_SIZE_2 	6	
#define DEFAULT_IMAGE	"Cross.pgm"

// Debug mode
//#define DEBUG  

// Kernels (in houghKernels.cu)
__global__ void sobel(u_char * frame_in, u_char * frame_out, int width, int height);
__global__ void houghTransform(u_char * frame_in, u_char * frame_out, const int hough_h);

// Global variables for RT threads
pthread_attr_t rt_sched_attr;
int rt_max_prio;
struct sched_param rt_param;
pid_t mainpid;
pthread_t rt_thread;

// Globals for Transform
unsigned int img_width;
unsigned int img_height;
unsigned int img_chan;
u_char* input_image;
u_char *result;
int hough_height;
int hough_width;
struct timespec run_time = {0, 0};
bool run_once = false;
int freq = 0;
std::string imageFilename = DEFAULT_IMAGE;

/***********************************************************
 * Functions to cleanup after code complete		  **
 ***********************************************************/
void Cleanup(void)
{
#ifdef DEBUG
	printf("DEBUG: Cleanup().\n");
#endif
     hipDeviceReset() ;
    exit(0);
}

//***************************************************************//
// Take the difference of two timespec structures
//***************************************************************//
void timespec_diff(struct timespec *start, struct timespec *stop,
                   struct timespec *result, bool check_neg)
{        
	result->tv_sec = stop->tv_sec - start->tv_sec;
        result->tv_nsec = stop->tv_nsec - start->tv_nsec;

    if ( check_neg && result->tv_nsec < 0) {
        result->tv_sec = result->tv_sec - 1;
        result->tv_nsec = result->tv_nsec + 1000000000;

    }
}

//***************************************************************//
// Convert timespec to double containing time in ms
//***************************************************************//
double timespec2double( struct timespec time_in)
{
	double rv;
	rv = (((double)time_in.tv_sec)*1000)+(((double)time_in.tv_nsec)/1000000);
	return rv;
}

//***************************************************************//
// Transform thread
//***************************************************************//
void *CUDA_transform_thread(void * threadp)
{
	// CUDA transform local variables
	struct timespec start_time, end_time, elap_time, diff_time;
	hipError_t errVal;
	double start_time_d, end_time_d, elap_time_d, diff_time_d;
	u_char* devInImage;
	u_char* devTresholded;
	u_char *A;
	
	// initialize needed variables
	int size = img_width*img_height;
	const int hough_h = (int) (sqrt(2.0) * img_width / 2.0f);
	hough_height = hough_h * 2;
	hough_width = 180;
	
	// Allocate memory for Hough output
    result = (u_char*) malloc(hough_height*hough_width * sizeof(u_char));

	dim3 dimBlock(BLOCK_SIZE_1, BLOCK_SIZE_2);
	dim3 dimGrid(img_width / dimBlock.x, img_height / dimBlock.y);

	printf("Filtering started...\n");
	// Allocate CUDA memory
	errVal = hipMalloc((void**)&devInImage, size*sizeof(u_char));
	if( errVal != hipSuccess)
		{ printf("hipMalloc error. %s\n",hipGetErrorString(errVal)); exit(-1); }
	errVal = hipMalloc((void**)&devTresholded, size*sizeof(u_char));
	if( errVal != hipSuccess)
		{ printf("hipMalloc error. %s\n",hipGetErrorString(errVal)); exit(-1); }
        errVal = hipMalloc((void**)&A,sizeof(u_char)*hough_height*hough_width);
	if( errVal != hipSuccess)
		{ printf("hipMalloc error. %s\n",hipGetErrorString(errVal)); exit(-1); }

	// loop to allow for power measurement
	do
	{
		// Get start of runtime timing
		if(clock_gettime(CLOCK_REALTIME, &start_time) )
		{
		  printf("clock_gettime() - start - error.. exiting.\n");
		  break;
		}
		start_time_d = timespec2double(start_time);

////////////////////////////////// BEGIN TRANSFORM ///////////////////////////////////
		errVal = hipMemcpy(devInImage, input_image, size*sizeof(u_char), hipMemcpyHostToDevice);
		if( errVal != hipSuccess)
			{ printf("cudaMemcpy1 error. %s\n",hipGetErrorString(errVal)); exit(-1); }

		// Complete the Sobel transform to find edges
		sobel<<<dimGrid,dimBlock>>>(devInImage, devTresholded,img_width,img_height);

		// Complete the Hough transform on the transformed sobel image
        houghTransform<<<dimGrid,dimBlock>>>(devTresholded, A, hough_h);
		
		hipDeviceSynchronize();

		hipMemcpy(result, A, hough_width*hough_height*sizeof(u_char),  hipMemcpyDeviceToHost);
		if( errVal != hipSuccess)
			{ printf("cudaMemcpy5 error. %s\n",hipGetErrorString(errVal)); exit(-1); }

#ifdef DEBUG
		printf("DEBUG: End transform.\n");
#endif
////////////////////////////////// END TRANSFORM ///////////////////////////////////
		// Get end of transform time timing
		if(clock_gettime(CLOCK_REALTIME, &end_time) )
		{
		  printf("clock_gettime() - end - error.. exiting.\n");
		  break;
		}
		
		if(run_time.tv_nsec != 0)
		{
			// Calculate the timing for nanosleep
			timespec_diff(&start_time, &end_time, &elap_time, true);
			timespec_diff(&elap_time, &run_time, &diff_time, false);
#ifdef DEBUG
			end_time_d = timespec2double(end_time);
			elap_time_d = end_time_d - start_time_d;
			diff_time_d = timespec2double(diff_time);
			printf("DEBUG: Transform runtime: %fms\n       Sleep time:       %fms\n",  elap_time_d, diff_time_d);
#endif	
			if(diff_time.tv_sec < 0 || diff_time.tv_nsec < 0)
			{
				diff_time_d = timespec2double(diff_time);
				printf("TIME OVERRUN by %fms---------\n", -diff_time_d);  
			} else
			{
				// Sleep for time needed to allow for running at known frequency 
				int err = nanosleep(&diff_time, &end_time);			
				if(err == -1)
				{
				   printf("\nFreq delay interrupted. Exiting..\n");
				   printf("**%d - %s**\n", errno, strerror(errno));
				   break;
				}
			}
		}

		// Get and calculate end of runtime time
		if(clock_gettime(CLOCK_REALTIME, &end_time) )
		{
		  printf("clock_gettime() - end - error.. exiting.\n");
		  break;
		}
		end_time_d = timespec2double(end_time);
		elap_time_d = end_time_d - start_time_d;
		printf("     Freq: %f Hz\n", 1000.0/elap_time_d);
	} while(!run_once);

	hipFree(devInImage);
	hipFree(devTresholded);

	return NULL; // to supress no return warnings.
}

//***************************************************************//
// Main function
//***************************************************************//
int main(int argc, char* argv[])
{
    // Local variables
	Options options(argc, argv);
	int errVal = 0;
	struct sched_param main_param;
	bool use_cuda = true;
	int tempInt, rv;
	char tempChar[25];

	// Check input
	if(options.has("help") || options.has("h") || options.has("?")) 
	{
		std::cout << "Usage: " << argv[0] << " [-continuous [-fps=FPS]] [-img=imageFilename] [-cuda] " << std::endl;
		exit(EXIT_SUCCESS);
	}
	
	if(options.has("img")) 
	{
		imageFilename = options.get<std::string>("img");
	}
	std::cout << "Img set to " << imageFilename << std::endl;

	if(options.has("continuous")) 
	{
		run_once = false;
		std::cout << "Continuous mode" << std::endl;
		if(options.has("fps")) 
		{
		  freq = options.get<unsigned int>("fps");
		  std::cout << "FPS Limiter set at " << freq << std::endl;
		} else 
		{
		  freq = 0;
		  std::cout << "FPS Unlimited" << std::endl;
		}
	} else 
	{
		run_once = true;
		std::cout << "Single shot mode" << std::endl;
	}

	if (options.has("cuda")) 
	{
		std::cout << "Program will use CUDA for transform." << std::endl;
		use_cuda = true;
	} else 
	{
		use_cuda = false;
		std::cout << "Program will use CPU for transform." << std::endl;
	}

#ifdef DEBUG
	printf("DEBUG: Begin Program. \n");
#endif	
	
	// Read Input image
	printf("Reading input image...");
	rv = parse_ppm_header((const char *) imageFilename.c_str(), &img_width, &img_height, &img_chan);
	if(!rv) 
	{
		printf("error reading file.\n"); 
		exit(-1); 
	}

	input_image = (unsigned char *)malloc(sizeof(unsigned int) * img_width * img_height);
	readppm(input_image, &tempInt, 
             tempChar, &tempInt,
             &img_height, &img_width, &img_chan,
             (char *)imageFilename.c_str());
	
	printf("\nWidth:%d  Height:%d\n",img_width, img_height);
	printf("[done]\n");
	
	// Pre-setup for Real-time threads
	mainpid = getpid();
	rt_max_prio = sched_get_priority_max(SCHED_FIFO);
	sched_getparam(mainpid, &main_param);
	main_param.sched_priority = rt_max_prio;
	errVal = sched_setscheduler(mainpid, SCHED_FIFO, &main_param);
	if(errVal < 0) 
		perror("main_param error");
		
	// Setup real-time thread
	pthread_attr_init(&rt_sched_attr);
	pthread_attr_setinheritsched(&rt_sched_attr, PTHREAD_EXPLICIT_SCHED);
	pthread_attr_setschedpolicy(&rt_sched_attr, SCHED_FIFO);
	rt_param.sched_priority=rt_max_prio-1;
	pthread_attr_setschedparam(&rt_sched_attr, &rt_param);

	if (freq) 
		run_time.tv_nsec = (1000000000/freq);
	else 
		run_time.tv_nsec = 0;

	// If continuous inform user that infinite loop will be entered to allow for power measurement
	if (run_once)
		printf("Program will now transform image once\n");
	else
		printf("Program will enter an infinite loop, use Ctrl+C to exit program when done.\n"); 
	
	printf("Press enter to proceed...");
	std::cin.ignore(); // Pause to allow user to read
	
	// Start Transform
	if(use_cuda)
    {					
		pthread_create(&rt_thread,   	// pointer to thread descriptor
			NULL,     		// use default attributes
			CUDA_transform_thread,	// thread function entry point
			&rt_param 		// parameters to pass in
			);
	}
	else // Use CPU version for transform
    {
		printf("No CPU version of Hough transform available. Please use CUDA by specifying '-cuda' in program call.\n");
	}	
	
	// Let transform thread run
	
	// Wait for thread to exit
	pthread_join(rt_thread, NULL);
	
	// Writeback results
	dump_ppm_data("hough.ppm", hough_width, hough_height, img_chan, result);
	
	// Free memory
	free(input_image);
	free(result);

	// Final Cleanup
	Cleanup();

    return 0;
}
